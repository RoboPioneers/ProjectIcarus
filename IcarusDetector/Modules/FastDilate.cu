#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <cmath>

__global__ void DeviceFastDilate(cv::cuda::PtrStepb input, cv::cuda::PtrStepb output,
                                 unsigned int picture_width, unsigned int picture_height,
                                 unsigned int kernel_width, unsigned int kernel_height,
                                 unsigned char lower_bound, unsigned char upper_bound,
                                 unsigned char target_value)
{
    const unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

    bool found = false;
    const unsigned int x_begin = id_x * kernel_width;
    const unsigned int y_begin = id_y * kernel_height;
    const unsigned int x_end = x_begin + kernel_width <= picture_width ? x_begin + kernel_width : picture_width;
    const unsigned int y_end = y_begin + kernel_height <= picture_height ? y_begin + kernel_height : picture_height;
    for (int row = static_cast<int>(y_begin); row < y_end; ++row)
    {
        for (int column = static_cast<int>(x_begin); column < x_end; ++column)
        {
            const unsigned char& value = input(row, column);
            if(value > lower_bound && value <= upper_bound)
            {
                found = true;
                break;
            }
        }
        if (found)
            break;
    }
    if (!found) return;
    for (int row = static_cast<int>(y_begin); row < y_end; ++row)
    {
        for (int column = static_cast<int>(x_begin); column < x_end; ++column)
        {
            output(row, column) = target_value;
        }
    }
}

__global__ void DeviceFastDilate32S(cv::cuda::PtrStepi input, cv::cuda::PtrStepi output,
                                 unsigned int picture_width, unsigned int picture_height,
                                 unsigned int kernel_width, unsigned int kernel_height,
                                 int lower_bound, int upper_bound,
                                 int target_value)
{
    const unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

    bool found = false;
    const unsigned int x_begin = id_x * kernel_width;
    const unsigned int y_begin = id_y * kernel_height;
    const unsigned int x_end = x_begin + kernel_width <= picture_width ? x_begin + kernel_width : picture_width;
    const unsigned int y_end = y_begin + kernel_height <= picture_height ? y_begin + kernel_height : picture_height;
    for (int row = static_cast<int>(y_begin); row < y_end; ++row)
    {
        for (int column = static_cast<int>(x_begin); column < x_end; ++column)
        {
            const int& value = input(row, column);
            if(value > lower_bound && value <= upper_bound)
            {
                found = true;
                break;
            }
        }
        if (found)
            break;
    }
    if (!found) return;
    for (int row = static_cast<int>(y_begin); row < y_end; ++row)
    {
        for (int column = static_cast<int>(x_begin); column < x_end; ++column)
        {
            output(row, column) = target_value;
        }
    }
}

/**
 * @brief Fast-Dilate is a fast algorithm to get the similar effect to dilate with a large kernel.
 * @param input
 * @param output
 * @param size
 * @param stream
 */
void FastDilate(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const cv::Size& area_size,
                unsigned char lower_bound, unsigned char upper_bound,
                unsigned int target_value, const cv::cuda::Stream& stream)
{
    if (output.empty())
    {
        output = cv::cuda::GpuMat(input.size(), CV_8UC1, cv::Scalar(0));
    }

    dim3 block_size;
    block_size.x = 16;
    block_size.y = 16;
    block_size.z = 1;
    dim3 grid_size;
    grid_size.x = std::ceil(input.cols * 1.0 / area_size.width / 16.0);
    grid_size.y = std::ceil(input.rows * 1.0 / area_size.height / 16.0);
    DeviceFastDilate<<<grid_size, block_size, 0, cv::cuda::StreamAccessor::getStream(stream)>>>(
            input, output, input.cols, input.rows, area_size.width, area_size.height,
            lower_bound, upper_bound, target_value);
}

/**
 * @brief Fast-Dilate is a fast algorithm to get the similar effect to dilate with a large kernel.
 * @param input
 * @param output
 * @param size
 * @param stream
 */
void FastDilate32S(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const cv::Size& area_size,
                int lower_bound, int upper_bound, int target_value, const cv::cuda::Stream& stream)
{
    if (output.empty())
    {
        output = cv::cuda::GpuMat(input.size(), CV_8UC1, cv::Scalar(0));
    }

    dim3 block_size;
    block_size.x = 16;
    block_size.y = 16;
    block_size.z = 1;
    dim3 grid_size;
    grid_size.x = std::ceil(input.cols * 1.0 / area_size.width / 16.0);
    grid_size.y = std::ceil(input.rows * 1.0 / area_size.height / 16.0);
    DeviceFastDilate32S<<<grid_size, block_size, 0, cv::cuda::StreamAccessor::getStream(stream)>>>(
            input, output, input.cols, input.rows, area_size.width, area_size.height,
            lower_bound, upper_bound, target_value);
}